#include "hip/hip_runtime.h"
#include "gtom/include/Prerequisites.cuh"
#include "gtom/include/Helper.cuh"

namespace gtom
{
	hipArray_t d_MallocArray(int2 dims)
	{
		hipChannelFormatDesc desc = hipCreateChannelDesc<tfloat>();
		hipArray_t a_input;
		hipMallocArray(&a_input, &desc, dims.x, dims.y);

		return a_input;
	}

	void d_MemcpyToArray(tfloat* d_input, hipArray_t a_output, int2 dims)
	{
		hipMemcpyToArray(a_output, 0, 0, d_input, dims.x * dims.y * sizeof(tfloat), hipMemcpyDeviceToDevice);
	}

	void d_BindTextureToArray(tfloat* d_input, hipArray_t &createdarray, cudaTex &createdtexture, int2 dims, hipTextureFilterMode filtermode, bool normalizedcoords)
	{
		hipChannelFormatDesc desc = hipCreateChannelDesc<tfloat>();
		hipArray* a_input;
		hipMallocArray(&a_input, &desc, dims.x, dims.y);
		hipMemcpyToArray(a_input, 0, 0, d_input, dims.x * dims.y * sizeof(tfloat), hipMemcpyDeviceToDevice);

		struct hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = a_input;

		struct hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(texDesc));
		texDesc.filterMode = filtermode;
		texDesc.readMode = hipReadModeElementType;
		texDesc.normalizedCoords = normalizedcoords;
		texDesc.addressMode[0] = hipAddressModeWrap;
		texDesc.addressMode[1] = hipAddressModeWrap;
		texDesc.addressMode[2] = hipAddressModeWrap;
		cudaTex texObj = 0;
		hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

		createdarray = a_input;
		createdtexture = texObj;
	}

	void d_BindTextureToArray(hipArray_t a_input, cudaTex& createdtexture, int2 dims, hipTextureFilterMode filtermode, bool normalizedcoords)
	{
		struct hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = a_input;

		struct hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(texDesc));
		texDesc.filterMode = filtermode;
		texDesc.readMode = hipReadModeElementType;
		texDesc.normalizedCoords = normalizedcoords;
		texDesc.addressMode[0] = hipAddressModeWrap;
		texDesc.addressMode[1] = hipAddressModeWrap;
		texDesc.addressMode[2] = hipAddressModeWrap;
		cudaTex texObj = 0;
		hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

		createdtexture = texObj;
	}

	void d_BindTextureToArray(tfloat* d_input, hipArray_t* &h_createdarrays, cudaTex* &h_createdtextures, int2 dims, hipTextureFilterMode filtermode, bool normalizedcoords, int nimages)
	{
		for (int n = 0; n < nimages; n++)
		{
			hipChannelFormatDesc desc = hipCreateChannelDesc<tfloat>();
			hipArray* a_input;
			hipMallocArray(&a_input, &desc, dims.x, dims.y);
			hipMemcpyToArray(a_input, 0, 0, d_input + Elements2(dims) * n, dims.x * dims.y * sizeof(tfloat), hipMemcpyDeviceToDevice);

			struct hipResourceDesc resDesc;
			memset(&resDesc, 0, sizeof(resDesc));
			resDesc.resType = hipResourceTypeArray;
			resDesc.res.array.array = a_input;

			struct hipTextureDesc texDesc;
			memset(&texDesc, 0, sizeof(texDesc));
			texDesc.filterMode = filtermode;
			texDesc.readMode = hipReadModeElementType;
			texDesc.normalizedCoords = normalizedcoords;
			texDesc.addressMode[0] = hipAddressModeWrap;
			texDesc.addressMode[1] = hipAddressModeWrap;
			texDesc.addressMode[2] = hipAddressModeWrap;
			cudaTex texObj = 0;
			hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

			h_createdarrays[n] = a_input;
			h_createdtextures[n] = texObj;
		}
	}

	void d_BindTextureTo3DArray(tfloat* d_input, hipArray_t &createdarray, cudaTex &createdtexture, int3 dims, hipTextureFilterMode filtermode, bool normalizedcoords)
	{
		hipChannelFormatDesc desc = hipCreateChannelDesc<tfloat>();
		hipArray* a_input;
		hipMalloc3DArray(&a_input, &desc, make_hipExtent(dims.x, dims.y, dims.z));

		hipMemcpy3DParms p = { 0 };
		p.extent = make_hipExtent(dims.x, dims.y, dims.z);
		p.srcPtr = make_hipPitchedPtr(d_input, dims.x * sizeof(tfloat), dims.x, dims.y);
		p.dstArray = a_input;
		p.kind = hipMemcpyDeviceToDevice;
		hipMemcpy3D(&p);

		struct hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(hipResourceDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = a_input;

		struct hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(hipTextureDesc));
		texDesc.filterMode = filtermode;
		texDesc.readMode = hipReadModeElementType;
		texDesc.normalizedCoords = normalizedcoords;
		texDesc.addressMode[0] = hipAddressModeWrap;
		texDesc.addressMode[1] = hipAddressModeWrap;
		texDesc.addressMode[2] = hipAddressModeWrap;
		cudaTex texObj = 0;
		hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

		createdarray = a_input;
		createdtexture = texObj;
	}

	void d_BindTextureTo3DArray(tfloat* d_input, hipArray_t* &h_createdarrays, cudaTex* &h_createdtextures, int3 dims, hipTextureFilterMode filtermode, bool normalizedcoords, int nvolumes)
	{
		for (int n = 0; n < nvolumes; n++)
		{
			hipChannelFormatDesc desc = hipCreateChannelDesc<tfloat>();
			hipArray* a_input;
			hipMalloc3DArray(&a_input, &desc, make_hipExtent(dims.x, dims.y, dims.z));

			hipMemcpy3DParms p = { 0 };
			p.extent = make_hipExtent(dims.x, dims.y, dims.z);
			p.srcPtr = make_hipPitchedPtr(d_input + Elements(dims) * n, dims.x * sizeof(tfloat), dims.x, dims.y);
			p.dstArray = a_input;
			p.kind = hipMemcpyDeviceToDevice;
			hipMemcpy3D(&p);

			struct hipResourceDesc resDesc;
			memset(&resDesc, 0, sizeof(resDesc));
			resDesc.resType = hipResourceTypeArray;
			resDesc.res.array.array = a_input;

			struct hipTextureDesc texDesc;
			memset(&texDesc, 0, sizeof(texDesc));
			texDesc.filterMode = filtermode;
			texDesc.readMode = hipReadModeElementType;
			texDesc.normalizedCoords = normalizedcoords;
			texDesc.addressMode[0] = hipAddressModeWrap;
			texDesc.addressMode[1] = hipAddressModeWrap;
			texDesc.addressMode[2] = hipAddressModeWrap;
			cudaTex texObj = 0;
			hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

			h_createdarrays[n] = a_input;
			h_createdtextures[n] = texObj;
		}
	}
}