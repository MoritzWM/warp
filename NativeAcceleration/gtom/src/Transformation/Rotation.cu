#include "hip/hip_runtime.h"
#include "gtom/include/Prerequisites.cuh"
#include "gtom/include/Angles.cuh"
#include "gtom/include/CubicInterp.cuh"
#include "gtom/include/DeviceFunctions.cuh"
#include "gtom/include/FFT.cuh"
#include "gtom/include/Generics.cuh"
#include "gtom/include/Helper.cuh"
#include "gtom/include/Transformation.cuh"


namespace gtom
{
	////////////////////////////
	//CUDA kernel declarations//
	////////////////////////////

	template<bool cubicinterp, bool outputzerocentered> __global__ void Rotate3DKernel(cudaTex t_input, tfloat* d_output, int3 dims, glm::mat4* d_transforms, uint nangles);
    template<bool cubicinterp> __global__ void Rotate3DExtractAt(cudaTex t_input, int3 dimvolume, tfloat* d_proj, int3 dimproj, size_t elementsproj, glm::mat3* d_rotations, tfloat3* d_positions);
	template<bool cubicinterp, bool outputzerocentered> __global__ void Rotate2DKernel(cudaTex* t_input, tfloat* d_output, int2 dims, glm::mat2* d_transforms);
	template<bool cubicinterp, bool outputzerocentered> __global__ void Rotate2DFTKernel(cudaTex t_Re, cudaTex t_Im, tcomplex* d_output, int3 dims, glm::mat2 transform, tfloat maxfreq);
	template<bool cubicinterp, bool outputzerocentered> __global__ void Rotate3DFTKernel(cudaTex t_Re, cudaTex t_Im, tcomplex* d_output, int3 dims, glm::mat4* d_transform, float maxfreq2);
	template<bool cubicinterp, bool outputzerocentered> __global__ void Rotate3DFTKernel(cudaTex t_Re, tfloat* d_output, int3 dims, glm::mat4* d_transform, float maxfreq2);


	////////////////////
	//Rotate 3D volume//
	////////////////////

	void d_Rotate3D(tfloat* d_volume, tfloat* d_output, int3 dims, tfloat3* h_angles, uint nangles, T_INTERP_MODE mode, bool iszerocentered)
	{
		tfloat* d_temp;
		if (mode == T_INTERP_CUBIC)
			hipMalloc((void**)&d_temp, Elements(dims) * sizeof(tfloat));

		hipArray* a_input;
		cudaTex t_input;
		if (mode == T_INTERP_LINEAR)
			d_BindTextureTo3DArray(d_volume, a_input, t_input, dims, hipFilterModeLinear, false);
		else
		{
			hipMemcpy(d_temp, d_volume, Elements(dims) * sizeof(tfloat), hipMemcpyDeviceToDevice);
			d_CubicBSplinePrefilter3D(d_temp, dims);
			d_BindTextureTo3DArray(d_temp, a_input, t_input, dims, hipFilterModeLinear, false);
		}

		d_Rotate3D(t_input, d_output, dims, h_angles, nangles, mode, iszerocentered);

		hipDestroyTextureObject(t_input);
		hipFreeArray(a_input);

		if (mode == T_INTERP_CUBIC)
			hipFree(d_temp);
	}

	void d_Rotate3D(cudaTex t_volume, tfloat* d_output, int3 dims, tfloat3* h_angles, uint nangles, T_INTERP_MODE mode, bool iszerocentered)
	{
		glm::mat4* h_transforms = (glm::mat4*)malloc(nangles * sizeof(glm::mat4));
		for (uint n = 0; n < nangles; n++)
		h_transforms[n] = Matrix4Translation(tfloat3(dims.x / 2 + 0.5f, dims.y / 2 + 0.5f, dims.z / 2 + 0.5f)) *
							glm::transpose(Matrix4Euler(h_angles[n])) *
							Matrix4Translation(tfloat3(-dims.x / 2, -dims.y / 2, -dims.z / 2));

		glm::mat4* d_transforms = (glm::mat4*)CudaMallocFromHostArray(h_transforms, nangles * sizeof(glm::mat4));
		free(h_transforms);

		dim3 TpB = dim3(16, 16);
		dim3 grid = dim3((dims.x + 15) / 16, (dims.y + 15) / 16, dims.z);

		if (iszerocentered)
		{
			if (mode == T_INTERP_LINEAR)
				Rotate3DKernel<false, true> << <grid, TpB >> > (t_volume, d_output, dims, d_transforms, nangles);
			else if (mode == T_INTERP_CUBIC)
				Rotate3DKernel<true, true> << <grid, TpB >> > (t_volume, d_output, dims, d_transforms, nangles);
		}
		else
		{
			if (mode == T_INTERP_LINEAR)
				Rotate3DKernel<false, false> << <grid, TpB >> > (t_volume, d_output, dims, d_transforms, nangles);
			else if (mode == T_INTERP_CUBIC)
				Rotate3DKernel<true, false> << <grid, TpB >> > (t_volume, d_output, dims, d_transforms, nangles);
		}

		hipFree(d_transforms);
	}


    ////////////////////////////
    //Rotate 3D and extract at//
    ////////////////////////////

    void d_Rotate3DExtractAt(cudaTex t_volume, int3 dimsvolume, tfloat* d_proj, int3 dimsproj, tfloat3* h_angles, tfloat3* h_positions, T_INTERP_MODE mode, uint batch)
    {
        glm::mat3* h_matrices = (glm::mat3*)malloc(sizeof(glm::mat3) * batch);
        for (int i = 0; i < batch; i++)
            h_matrices[i] = glm::transpose(Matrix3Euler(h_angles[i]));
        glm::mat3* d_matrices = (glm::mat3*)CudaMallocFromHostArray(h_matrices, sizeof(glm::mat3) * batch);
        free(h_matrices);

        tfloat3* d_positions = (tfloat3*)CudaMallocFromHostArray(h_positions, batch * sizeof(tfloat3));

        d_Rotate3DExtractAt(t_volume, dimsvolume, d_proj, dimsproj, d_matrices, d_positions, mode, batch);

        hipFree(d_matrices);
        hipFree(d_positions);
    }

    void d_Rotate3DExtractAt(cudaTex t_volume, int3 dimsvolume, tfloat* d_proj, int3 dimsproj, glm::mat3* d_matrices, tfloat3* d_positions, T_INTERP_MODE mode, uint batch)
    {
        uint ndimsvolume = DimensionCount(dimsvolume);
        uint ndimsproj = DimensionCount(dimsproj);
        if (ndimsvolume < ndimsproj)
            throw;

        uint elements = Elements(dimsproj);
        dim3 grid = dim3(tmin(128, (elements + 127) / 128), batch, 1);

        if (ndimsproj >= 2)
        {
            if (mode == T_INTERP_CUBIC)
                Rotate3DExtractAt<true> << <grid, 128 >> > (t_volume, dimsvolume, d_proj, dimsproj, elements, d_matrices, d_positions);
            else
                Rotate3DExtractAt<false> << <grid, 128 >> > (t_volume, dimsvolume, d_proj, dimsproj, elements, d_matrices, d_positions);
        }
        else
            throw;
    }


	/////////////
	//Rotate 2D//
	/////////////
	
	void d_Rotate2D(tfloat* d_input, tfloat* d_output, int2 dims, tfloat* h_angles, T_INTERP_MODE mode, bool isoutputzerocentered, uint batch)
	{
		tfloat* d_temp;
		if (mode == T_INTERP_CUBIC)
			hipMalloc((void**)&d_temp, Elements2(dims) * batch * sizeof(tfloat));

		hipArray_t* a_input = (hipArray_t*)malloc(batch * sizeof(hipArray_t));
		cudaTex* t_input = (cudaTex*)malloc(batch * sizeof(cudaTex));
		if (mode == T_INTERP_LINEAR)
			d_BindTextureToArray(d_input, a_input, t_input, dims, hipFilterModeLinear, false, batch);
		else
		{
			hipMemcpy(d_temp, d_input, Elements2(dims) * batch * sizeof(tfloat), hipMemcpyDeviceToDevice);
			d_CubicBSplinePrefilter2D(d_temp, dims, batch);
			d_BindTextureToArray(d_temp, a_input, t_input, dims, hipFilterModeLinear, false, batch);
		}
		cudaTex* dt_input = (cudaTex*)CudaMallocFromHostArray(t_input, batch * sizeof(cudaTex));

		d_Rotate2D(dt_input, d_output, dims, h_angles, mode, isoutputzerocentered, batch);

		hipFree(dt_input);
		for (uint b = 0; b < batch; b++)
		{
			hipDestroyTextureObject(t_input[b]);
			hipFreeArray(a_input[b]);
		}
		free(t_input);
		free(a_input);
		if (mode == T_INTERP_CUBIC)
			hipFree(d_temp);
	}

	void d_Rotate2D(cudaTex* t_input, tfloat* d_output, int2 dims, tfloat* h_angles, T_INTERP_MODE mode, bool isoutputzerocentered, uint batch)
	{
		glm::mat2* h_transforms = (glm::mat2*)malloc(batch * sizeof(glm::mat2));
		for (uint b = 0; b < batch; b++)
			h_transforms[b] = Matrix2Rotation(-h_angles[b]);
		glm::mat2* d_transforms = (glm::mat2*)CudaMallocFromHostArray(h_transforms, batch * sizeof(glm::mat2));
		free(h_transforms);

		dim3 TpB = dim3(16, 16);
		dim3 grid = dim3((dims.x + 15) / 16, (dims.y + 15) / 16, batch);

		if (isoutputzerocentered)
		{
			if (mode == T_INTERP_LINEAR)
				Rotate2DKernel<false, true> << <grid, TpB >> > (t_input, d_output, dims, d_transforms);
			else if (mode == T_INTERP_CUBIC)
				Rotate2DKernel<true, true> << <grid, TpB >> > (t_input, d_output, dims, d_transforms);
		}
		else
		{
			if (mode == T_INTERP_LINEAR)
				Rotate2DKernel<false, false> << <grid, TpB >> > (t_input, d_output, dims, d_transforms);
			else if (mode == T_INTERP_CUBIC)
				Rotate2DKernel<true, false> << <grid, TpB >> > (t_input, d_output, dims, d_transforms);
		}

		hipFree(d_transforms);
	}


	//////////////////////////////
	//Rotate 2D in Fourier space//
	//////////////////////////////

	void d_Rotate2DFT(tcomplex* d_input, tcomplex* d_output, int3 dims, tfloat* angles, tfloat maxfreq, T_INTERP_MODE mode, bool isoutputzerocentered, int batch)
	{
		tfloat* d_real;
		hipMalloc((void**)&d_real, ElementsFFT(dims) * sizeof(tfloat));
		tfloat* d_imag;
		hipMalloc((void**)&d_imag, ElementsFFT(dims) * sizeof(tfloat));

		for (int b = 0; b < batch; b++)
		{
			d_ConvertTComplexToSplitComplex(d_input + ElementsFFT(dims) * b, d_real, d_imag, ElementsFFT(dims));

			if (mode == T_INTERP_CUBIC)
			{
				d_CubicBSplinePrefilter2D(d_real, toInt2(dims.x / 2 + 1, dims.y));
				d_CubicBSplinePrefilter2D(d_imag, toInt2(dims.x / 2 + 1, dims.y));
			}

			hipArray* a_Re;
			hipArray* a_Im;
			cudaTex t_Re, t_Im;
			d_BindTextureToArray(d_real, a_Re, t_Re, toInt2(dims.x / 2 + 1, dims.y), hipFilterModeLinear, false);
			d_BindTextureToArray(d_imag, a_Im, t_Im, toInt2(dims.x / 2 + 1, dims.y), hipFilterModeLinear, false);

			d_Rotate2DFT(t_Re, t_Im, d_output + ElementsFFT(dims) * b, dims, angles[b], maxfreq, mode, isoutputzerocentered);

			hipDestroyTextureObject(t_Re);
			hipDestroyTextureObject(t_Im);
			hipFreeArray(a_Re);
			hipFreeArray(a_Im);
		}

		hipFree(d_imag);
		hipFree(d_real);
	}

	void d_Rotate2DFT(cudaTex t_inputRe, cudaTex t_inputIm, tcomplex* d_output, int3 dims, tfloat angle, tfloat maxfreq, T_INTERP_MODE mode, bool isoutputzerocentered)
	{
		glm::mat2 rotation = Matrix2Rotation(-angle);

		dim3 TpB = dim3(16, 16);
		dim3 grid = dim3((dims.x / 2 + 1 + 15) / 16, (dims.y + 15) / 16);

		if (isoutputzerocentered)
		{
			if (mode == T_INTERP_LINEAR)
				Rotate2DFTKernel<false, true> << <grid, TpB >> > (t_inputRe, t_inputIm, d_output, dims, rotation, maxfreq);
			else if (mode == T_INTERP_CUBIC)
				Rotate2DFTKernel<true, true> << <grid, TpB >> > (t_inputRe, t_inputIm, d_output, dims, rotation, maxfreq);
		}
		else
		{
			if (mode == T_INTERP_LINEAR)
				Rotate2DFTKernel<false, false> << <grid, TpB >> > (t_inputRe, t_inputIm, d_output, dims, rotation, maxfreq);
			else if (mode == T_INTERP_CUBIC)
				Rotate2DFTKernel<true, false> << <grid, TpB >> > (t_inputRe, t_inputIm, d_output, dims, rotation, maxfreq);
		}
	}
	

	//////////////////////////////
	//Rotate 3D in Fourier space//
	//////////////////////////////

	void d_Rotate3DFT(tcomplex* d_volume, tcomplex* d_output, int3 dims, tfloat3* h_angles, int nangles, T_INTERP_MODE mode, bool outputzerocentered)
	{
		int3 dimsfft = toInt3(dims.x / 2 + 1, dims.y, dims.z);
		tfloat* d_tempRe;
		hipMalloc((void**)&d_tempRe, ElementsFFT(dims) * sizeof(tfloat));
		tfloat* d_tempIm;
		hipMalloc((void**)&d_tempIm, ElementsFFT(dims) * sizeof(tfloat));

		hipArray* a_Re, *a_Im;
		cudaTex t_Re, t_Im;

		d_ConvertTComplexToSplitComplex(d_volume, d_tempRe, d_tempIm, ElementsFFT(dims));
		if (mode == T_INTERP_CUBIC)
		{
			d_CubicBSplinePrefilter3D(d_tempRe, dimsfft);
			d_CubicBSplinePrefilter3D(d_tempIm, dimsfft);
		}
		d_BindTextureTo3DArray(d_tempRe, a_Re, t_Re, dimsfft, hipFilterModeLinear, false);
		d_BindTextureTo3DArray(d_tempIm, a_Im, t_Im, dimsfft, hipFilterModeLinear, false);
		hipFree(d_tempRe);
		hipFree(d_tempIm);

		d_Rotate3DFT(t_Re, t_Im, d_output, dims, h_angles, nangles, mode, outputzerocentered);

		hipDestroyTextureObject(t_Re);
		hipDestroyTextureObject(t_Im);
		hipFreeArray(a_Re);
		hipFreeArray(a_Im);
	}

	void d_Rotate3DFT(cudaTex t_Re, cudaTex t_Im, tcomplex* d_output, int3 dims, tfloat3* h_angles, int nangles, T_INTERP_MODE mode, bool outputzerocentered)
	{
		glm::mat4* h_transform = (glm::mat4*)malloc(nangles * sizeof(glm::mat4));
		for (int b = 0; b < nangles; b++)
			h_transform[b] = glm::transpose(Matrix4Euler(h_angles[b])) *
							 Matrix4Translation(tfloat3(-dims.x / 2, -dims.y / 2, -dims.z / 2));
		glm::mat4* d_transform = (glm::mat4*)CudaMallocFromHostArray(h_transform, nangles * sizeof(glm::mat4));

		float maxfreq2 = (float)(dims.x * dims.x / 4);

		dim3 TpB = dim3(16, 16);
		dim3 grid = dim3((dims.x / 2 + 1 + 15) / 16, (dims.y + 15) / 16, dims.z * nangles);
		if (outputzerocentered)
		{
			if (mode == T_INTERP_LINEAR)
				Rotate3DFTKernel<false, true> << <grid, TpB >> > (t_Re, t_Im, d_output, dims, d_transform, maxfreq2);
			if (mode == T_INTERP_CUBIC)
				Rotate3DFTKernel<true, true> << <grid, TpB >> > (t_Re, t_Im, d_output, dims, d_transform, maxfreq2);
		}
		else
		{
			if (mode == T_INTERP_LINEAR)
				Rotate3DFTKernel<false, false> << <grid, TpB >> > (t_Re, t_Im, d_output, dims, d_transform, maxfreq2);
			if (mode == T_INTERP_CUBIC)
				Rotate3DFTKernel<true, false> << <grid, TpB >> > (t_Re, t_Im, d_output, dims, d_transform, maxfreq2);
		}

		hipFree(d_transform);
		free(h_transform);
	}

	void d_Rotate3DFT(tfloat* d_volume, tfloat* d_output, int3 dims, tfloat3* h_angles, int nangles, T_INTERP_MODE mode, bool outputzerocentered)
	{
		int3 dimsfft = toInt3(dims.x / 2 + 1, dims.y, dims.z);
		tfloat* d_tempRe;
		hipMalloc((void**)&d_tempRe, ElementsFFT(dims) * sizeof(tfloat));

		hipArray* a_Re;
		cudaTex t_Re;

		hipMemcpy(d_tempRe, d_volume, ElementsFFT(dims) * sizeof(tfloat), hipMemcpyDeviceToDevice);
		if (mode == T_INTERP_CUBIC)
			d_CubicBSplinePrefilter3D(d_tempRe, dimsfft);
		d_BindTextureTo3DArray(d_tempRe, a_Re, t_Re, dimsfft, hipFilterModeLinear, false);
		hipFree(d_tempRe);

		d_Rotate3DFT(t_Re, d_output, dims, h_angles, nangles, mode, outputzerocentered);

		hipDestroyTextureObject(t_Re);
		hipFreeArray(a_Re);
	}

	void d_Rotate3DFT(cudaTex t_volume, tfloat* d_output, int3 dims, tfloat3* h_angles, int nangles, T_INTERP_MODE mode, bool outputzerocentered)
	{
		glm::mat4* h_transform = (glm::mat4*)malloc(nangles * sizeof(glm::mat4));
		for (int b = 0; b < nangles; b++)
			h_transform[b] = glm::transpose(Matrix4Euler(h_angles[b])) *
							 Matrix4Translation(tfloat3(-dims.x / 2, -dims.y / 2, -dims.z / 2));
		glm::mat4* d_transform = (glm::mat4*)CudaMallocFromHostArray(h_transform, nangles * sizeof(glm::mat4));

		float maxfreq2 = (float)(dims.x * dims.x / 4);

		dim3 TpB = dim3(16, 16);
		dim3 grid = dim3((dims.x / 2 + 1 + 15) / 16, (dims.y + 15) / 16, dims.z * nangles);
		if (outputzerocentered)
		{
			if (mode == T_INTERP_LINEAR)
				Rotate3DFTKernel<false, true> << <grid, TpB >> > (t_volume, d_output, dims, d_transform, maxfreq2);
			if (mode == T_INTERP_CUBIC)
				Rotate3DFTKernel<true, true> << <grid, TpB >> > (t_volume, d_output, dims, d_transform, maxfreq2);
		}
		else
		{
			if (mode == T_INTERP_LINEAR)
				Rotate3DFTKernel<false, false> << <grid, TpB >> > (t_volume, d_output, dims, d_transform, maxfreq2);
			if (mode == T_INTERP_CUBIC)
				Rotate3DFTKernel<true, false> << <grid, TpB >> > (t_volume, d_output, dims, d_transform, maxfreq2);
		}

		hipFree(d_transform);
		free(h_transform);
	}


	////////////////
	//CUDA kernels//
	////////////////

	template<bool cubicinterp, bool outputzerocentered> __global__ void Rotate3DKernel(cudaTex t_input, tfloat* d_output, int3 dims, glm::mat4* d_transforms, uint nangles)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx >= dims.x)
			return;
		int idy = blockIdx.y * blockDim.y + threadIdx.y;
		if (idy >= dims.y)
			return;
		int idz = blockIdx.z;

		int x, y, z;
		if (outputzerocentered)
		{
			x = idx;
			y = idy;
			z = idz;
		}
		else
		{
			x = FFTShift(idx, dims.x);
			y = FFTShift(idy, dims.y);
			z = FFTShift(idz, dims.z);
		}

		for (uint b = 0; b < nangles; b++)
		{
			glm::vec4 pos = d_transforms[b] * glm::vec4(x, y, z, 1);	// No need to center pos, done by transform
			tfloat value;

			if (cubicinterp)
				value = cubicTex3DSimple<tfloat>(t_input, pos.x, pos.y, pos.z);
			else
				value = tex3D<tfloat>(t_input, pos.x, pos.y, pos.z);

			d_output[(b * dims.z + (idz * dims.y + idy)) * dims.x + idx] = value;
		}
	}

    template<bool cubicinterp> __global__ void Rotate3DExtractAt(cudaTex t_input, int3 dimvolume, tfloat* d_proj, int3 dimproj, size_t elementsproj, glm::mat3* d_rotations, tfloat3* d_positions)
    {
        d_proj += elementsproj * blockIdx.y;

        uint line = dimproj.x;
        uint slice = Elements2(dimproj);

        glm::mat3 rotation = d_rotations[blockIdx.y];
        glm::vec3 position = glm::vec3(d_positions[blockIdx.y].x, d_positions[blockIdx.y].y, d_positions[blockIdx.y].z);
        int3 centervolume = dimproj / 2;

        for (uint id = blockIdx.x * blockDim.x + threadIdx.x; id < elementsproj; id += gridDim.x * blockDim.x)
        {
            uint idx = id % line;
            uint idy = (id % slice) / line;
            uint idz = id / slice;

            int x = idx;
            int y = idy;
            int z = idz;

            glm::vec3 pos = glm::vec3(x - centervolume.x, y - centervolume.y, z - centervolume.z);

            pos = rotation * pos;
            pos += position;

			if (pos.x >= 0 && pos.y >= 0 && pos.z >= 0 &&
				pos.x <= dimvolume.x - 1 && pos.y <= dimvolume.y - 1 && pos.z <= dimvolume.z - 1)
			{
				pos += 0.5f;

				if (cubicinterp)
					d_proj[id] = cubicTex3DSimple<tfloat>(t_input, pos.x, pos.y, pos.z);
				else
					d_proj[id] = tex3D<tfloat>(t_input, pos.x, pos.y, pos.z);
			}
			else
			{
				glm::vec3 posclamped = glm::vec3(tmax(tmin(dimvolume.x - 1, pos.x), 0),
												 tmax(tmin(dimvolume.y - 1, pos.y), 0),
												 tmax(tmin(dimvolume.z - 1, pos.z), 0));
				float borderdist = glm::distance(pos, posclamped);
				
				if (borderdist > 16)
				{
					d_proj[id] = 0;
					continue;
				}

				int samples = 0;
				tfloat blursum = 0;

				for (int z = -3; z <= 3; z++)
				{
					for (int y = -3; y <= 3; y++)
					{
						for (int x = -3; x <= 3; x++)
						{
							glm::vec3 blurpos = posclamped + glm::vec3(x, y, z) / 3.0f * tmin(borderdist * 0.5f, 8);

							if (blurpos.x < -0.1f || blurpos.y < -0.1f || blurpos.z < -0.1f ||
								blurpos.x >= dimvolume.x - 0.9f || blurpos.y >= dimvolume.y - 0.9f || blurpos.z >= dimvolume.z - 0.9f)
								continue;

							samples++;
							blurpos += 0.5f;

							blursum += tex3D<tfloat>(t_input, blurpos.x, blurpos.y, blurpos.z);
						}
					}
				}

				float weight = cosf(borderdist / 16.0f * PI) * 0.5f + 0.5f;
				d_proj[id] = blursum / tmax(1, samples) * weight;
			}
        }
    }

	template<bool cubicinterp, bool outputzerocentered> __global__ void Rotate2DKernel(cudaTex* t_input, tfloat* d_output, int2 dims, glm::mat2* d_transforms)
	{
		uint idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx >= dims.x)
			return;
		uint idy = blockIdx.y * blockDim.y + threadIdx.y;
		if (idy >= dims.y)
			return;

		int x, y;
		if (outputzerocentered)
		{
			x = idx;
			y = idy;
		}
		else
		{
			x = FFTShift(idx, dims.x);
			y = FFTShift(idy, dims.y);
		}

		glm::vec2 pos = d_transforms[blockIdx.z] * glm::vec2(x - dims.x / 2, y - dims.y / 2) + glm::vec2(dims.x / 2 + 0.5f, dims.y / 2 + 0.5f);
		tfloat val;

		if (!cubicinterp)
			val = tex2D<tfloat>(t_input[blockIdx.z], pos.x, pos.y);
		else
			val = cubicTex2D(t_input[blockIdx.z], pos.x, pos.y);

		d_output[(blockIdx.z * dims.y + idy) * dims.x + idx] = val;
	}

	template<bool cubicinterp, bool outputzerocentered> __global__ void Rotate2DFTKernel(cudaTex t_Re, cudaTex t_Im, tcomplex* d_output, int3 dims, glm::mat2 transform, tfloat maxfreq)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx > dims.x / 2)
			return;
		int idy = blockIdx.y * blockDim.y + threadIdx.y;
		if (idy >= dims.y)
			return;

		int x, y;
		if (outputzerocentered)
		{
			x = idx;
			y = idy;
		}
		else
		{
			x = dims.x / 2 - idx;
			y = dims.y - 1 - ((idy + dims.y / 2 - 1) % dims.y);
		}

		glm::vec2 pos = transform * glm::vec2(idx - dims.x / 2, idy - dims.y / 2);

		if (glm::length(pos) > maxfreq)
		{
			d_output[y * (dims.x / 2 + 1) + x] = make_hipComplex(0.0f, 0.0f);
			return;
		}

		bool isnegative = false;
		if (pos.x > 0.00001f)
		{
			pos = -pos;
			isnegative = true;
		}

		pos += glm::vec2((float)(dims.x / 2) + 0.5f, (float)(dims.y / 2) + 0.5f);

		tfloat valre, valim;
		if (!cubicinterp)
		{
			valre = tex2D<tfloat>(t_Re, pos.x, pos.y);
			valim = tex2D<tfloat>(t_Im, pos.x, pos.y);
		}
		else
		{
			valre = cubicTex2D(t_Re, pos.x, pos.y);
			valim = cubicTex2D(t_Im, pos.x, pos.y);
		}

		if (isnegative)
			valim = -valim;

		d_output[y * (dims.x / 2 + 1) + x] = make_hipComplex(valre, valim);
	}

	template<bool cubicinterp, bool outputzerocentered> __global__ void Rotate3DFTKernel(cudaTex t_Re, cudaTex t_Im, tcomplex* d_output, int3 dims, glm::mat4* d_transform, float maxfreq2)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx > dims.x / 2)
			return;
		uint idglobal = blockIdx.z / dims.z;
		d_output += ElementsFFT(dims) * idglobal;
		d_transform += idglobal;

		int idy = blockIdx.y * blockDim.y + threadIdx.y;
		if (idy >= dims.y)
			return;
		int idz = blockIdx.z % dims.z;

		int x, y, z;
		if (outputzerocentered)
		{
			x = idx;
			y = idy;
			z = idz;
		}
		else
		{
			x = dims.x / 2 - idx;
			y = dims.y - 1 - ((idy + dims.y / 2 - 1) % dims.y);
			z = dims.z - 1 - ((idz + dims.z / 2 - 1) % dims.z);
		}

		glm::vec4 pos = *d_transform * glm::vec4(x, y, z, 1);

		float radiussq = pos.x * pos.x + pos.y * pos.y + pos.z * pos.z;
		if (radiussq >= maxfreq2)
		{
			d_output[(idz * dims.y + idy) * (dims.x / 2 + 1) + idx] = make_hipComplex(0, 0);
			return;
		}

		bool isnegative = false;
		if (pos.x > 1e-6f)
		{
			pos = -pos;
			isnegative = true;
		}

		pos += (float)(dims.x / 2) + 0.5f;

		tfloat valre, valim;
		if (!cubicinterp)
		{
			valre = tex3D<tfloat>(t_Re, pos.x, pos.y, pos.z);
			valim = tex3D<tfloat>(t_Im, pos.x, pos.y, pos.z);
		}
		else
		{
			valre = cubicTex3D(t_Re, pos.x, pos.y, pos.z);
			valim = cubicTex3D(t_Im, pos.x, pos.y, pos.z);
		}

		if (isnegative)
			valim = -valim;

		d_output[(idz * dims.y + idy) * (dims.x / 2 + 1) + idx] = make_hipComplex(valre, valim);
	}

	template<bool cubicinterp, bool outputzerocentered> __global__ void Rotate3DFTKernel(cudaTex t_Re, tfloat* d_output, int3 dims, glm::mat4* d_transform, float maxfreq2)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx > dims.x / 2)
			return;
		uint idglobal = blockIdx.z / dims.z;
		d_output += ElementsFFT(dims) * idglobal;
		d_transform += idglobal;

		int idy = blockIdx.y * blockDim.y + threadIdx.y;
		if (idy >= dims.y)
			return;
		int idz = blockIdx.z % dims.z;

		int x, y, z;
		if (outputzerocentered)
		{
			x = idx;
			y = idy;
			z = idz;
		}
		else
		{
			x = dims.x / 2 - idx;
			y = dims.y - 1 - ((idy + dims.y / 2 - 1) % dims.y);
			z = dims.z - 1 - ((idz + dims.z / 2 - 1) % dims.z);
		}

		glm::vec4 pos = *d_transform * glm::vec4(x, y, z, 1);

		float radiussq = pos.x * pos.x + pos.y * pos.y + pos.z * pos.z;
		if (radiussq >= maxfreq2)
		{
			d_output[(idz * dims.y + idy) * (dims.x / 2 + 1) + idx] = 0;
			return;
		}

		if (pos.x > 1e-6f)
			pos = -pos;

		pos += (float)(dims.x / 2) + 0.5f;

		tfloat valre;
		if (!cubicinterp)
			valre = tex3D<tfloat>(t_Re, pos.x, pos.y, pos.z);
		else
			valre = cubicTex3D(t_Re, pos.x, pos.y, pos.z);

		d_output[(idz * dims.y + idy) * (dims.x / 2 + 1) + idx] = valre;
	}
}