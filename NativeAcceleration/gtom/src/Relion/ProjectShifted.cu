#include "hip/hip_runtime.h"
#include "gtom/include/Prerequisites.cuh"
#include "gtom/include/Angles.cuh"
#include "gtom/include/Helper.cuh"
#include "gtom/include/Relion.cuh"

namespace gtom
{
	template<uint ndims, uint TpB> __global__ void ProjectShifted3DArraytoNDKernel(tcomplex* d_volume, 
																					uint dimvolume, 
																					tcomplex* d_proj, 
																					uint dimproj, 
																					size_t elementsproj, 
																					glm::mat3* d_rotations, 
																					tfloat3* d_shifts,
																					float* d_globalweights,
																					uint rmax, 
																					int rmax2);

	template<uint ndims, uint TpB> __global__ void ProjectShifted3DtoNDKernel(cudaTex t_volumeRe,
																				cudaTex t_volumeIm,
																				uint dimvolume,
																				tcomplex* d_proj,
																				uint dimproj,
																				size_t elementsproj,
																				glm::mat3* d_rotations,
																				tfloat3* d_shifts,
																				float* d_globalweights,
																				uint rmax,
																				int rmax2);


	void d_rlnProjectShifted(tcomplex* d_volumeft, int3 dimsvolume, tcomplex* d_proj, int3 dimsproj, tfloat3* h_angles, tfloat3* h_shifts, float* h_globalweights, float supersample, uint batch)
	{
		glm::mat3* h_matrices = (glm::mat3*)malloc(sizeof(glm::mat3) * batch);
		for (int i = 0; i < batch; i++)
			h_matrices[i] = glm::transpose(Matrix3Euler(h_angles[i])) * Matrix3Scale(supersample);
		glm::mat3* d_matrices = (glm::mat3*)CudaMallocFromHostArray(h_matrices, sizeof(glm::mat3) * batch);
		free(h_matrices);

		tfloat3* h_shiftsscaled = (tfloat3*)malloc(batch * sizeof(tfloat3));
		for (int i = 0; i < batch; i++)
			h_shiftsscaled[i] = tfloat3(h_shifts[i].x * PI2 / dimsproj.x,
										h_shifts[i].y * PI2 / dimsproj.x,
										h_shifts[i].z * PI2 / dimsproj.x);
		tfloat3* d_shifts = (tfloat3*)CudaMallocFromHostArray(h_shiftsscaled, batch * sizeof(tfloat3));
		free(h_shiftsscaled);

		float* d_globalweights = (float*)CudaMallocFromHostArray(h_globalweights, batch * sizeof(float));

		d_rlnProjectShifted(d_volumeft, dimsvolume, d_proj, dimsproj, dimsproj.x / 2, d_matrices, d_shifts, d_globalweights, batch);

		hipFree(d_globalweights);
		hipFree(d_shifts);
		hipFree(d_matrices);
	}

	void d_rlnProjectShifted(tcomplex* d_volumeft, int3 dimsvolume, tcomplex* d_proj, int3 dimsproj, uint rmax, glm::mat3* d_matrices, tfloat3* d_shifts, float* d_globalweights, uint batch)
	{
		uint ndimsvolume = DimensionCount(dimsvolume);
		uint ndimsproj = DimensionCount(dimsproj);
		if (ndimsvolume < ndimsproj)
			throw;

		rmax = tmin(rmax, dimsproj.x / 2);

		uint elements = ElementsFFT(dimsproj);
		dim3 grid = dim3(tmin(64, (elements + 127) / 128), batch, 1);

		if (ndimsproj == 2)
			ProjectShifted3DArraytoNDKernel<2, 128> << <grid, 128 >> > (d_volumeft, dimsvolume.x, d_proj, dimsproj.x, elements, d_matrices, d_shifts, d_globalweights, rmax, rmax * rmax);
		else if (ndimsproj == 3)
			ProjectShifted3DArraytoNDKernel<3, 128> << <grid, 128 >> > (d_volumeft, dimsvolume.x, d_proj, dimsproj.x, elements, d_matrices, d_shifts, d_globalweights, rmax, rmax * rmax);
		else
			throw;
	}


	void d_rlnProjectShifted(cudaTex t_volumeRe, cudaTex t_volumeIm, int3 dimsvolume, tcomplex* d_proj, int3 dimsproj, tfloat3* h_angles, tfloat3* h_shifts, float* h_globalweights, float supersample, uint batch)
	{
		glm::mat3* h_matrices = (glm::mat3*)malloc(sizeof(glm::mat3) * batch);
		for (int i = 0; i < batch; i++)
			h_matrices[i] = glm::transpose(Matrix3Euler(h_angles[i])) * Matrix3Scale(supersample);
		glm::mat3* d_matrices = (glm::mat3*)CudaMallocFromHostArray(h_matrices, sizeof(glm::mat3) * batch);
		free(h_matrices);

		tfloat3* h_shiftsscaled = (tfloat3*)malloc(batch * sizeof(tfloat3));
		for (int i = 0; i < batch; i++)
			h_shiftsscaled[i] = tfloat3(h_shifts[i].x * PI2 / dimsproj.x,
			h_shifts[i].y * PI2 / dimsproj.x,
			h_shifts[i].z * PI2 / dimsproj.x);
		tfloat3* d_shifts = (tfloat3*)CudaMallocFromHostArray(h_shiftsscaled, batch * sizeof(tfloat3));
		free(h_shiftsscaled);

		float* d_globalweights = (float*)CudaMallocFromHostArray(h_globalweights, batch * sizeof(float));

		d_rlnProjectShifted(t_volumeRe, t_volumeIm, dimsvolume, d_proj, dimsproj, dimsproj.x / 2, d_matrices, d_shifts, d_globalweights, batch);

		hipFree(d_globalweights);
		hipFree(d_shifts);
		hipFree(d_matrices);
	}

	void d_rlnProjectShifted(cudaTex t_volumeRe, cudaTex t_volumeIm, int3 dimsvolume, tcomplex* d_proj, int3 dimsproj, uint rmax, glm::mat3* d_matrices, tfloat3* d_shifts, float* d_globalweights, uint batch)
	{
		uint ndimsvolume = DimensionCount(dimsvolume);
		uint ndimsproj = DimensionCount(dimsproj);
		if (ndimsvolume < ndimsproj)
			throw;

		rmax = tmin(rmax, dimsproj.x / 2);

		uint elements = ElementsFFT(dimsproj);
		dim3 grid = dim3(tmin(64, (elements + 127) / 128), batch, 1);

		if (ndimsproj == 2)
			ProjectShifted3DtoNDKernel<2, 128> << <grid, 128 >> > (t_volumeRe, t_volumeIm, dimsvolume.x, d_proj, dimsproj.x, elements, d_matrices, d_shifts, d_globalweights, rmax, rmax * rmax);
		else if (ndimsproj == 3)
			ProjectShifted3DtoNDKernel<3, 128> << <grid, 128 >> > (t_volumeRe, t_volumeIm, dimsvolume.x, d_proj, dimsproj.x, elements, d_matrices, d_shifts, d_globalweights, rmax, rmax * rmax);
		else
			throw;
	}

	template<uint ndims, uint TpB> __global__ void ProjectShifted3DArraytoNDKernel(tcomplex* d_volume, 
																					uint dimvolume, 
																					tcomplex* d_proj, 
																					uint dimproj, 
																					size_t elementsproj, 
																					glm::mat3* d_rotations, 
																					tfloat3* d_shifts,
																					float* d_globalweights,
																					uint rmax, 
																					int rmax2)
	{
		d_proj += elementsproj * blockIdx.y;

		int x0, x1, y0, y1, z0, z1;
		tcomplex d000, d010, d100, d110, d001, d011, d101, d111, dx00, dx10, dxy0, dx01, dx11, dxy1;

		uint slice = ElementsFFT1(dimproj) * dimproj;
		uint dimft = ElementsFFT1(dimproj);

		glm::mat3 rotation = d_rotations[blockIdx.y];
		tfloat3 shift = d_shifts[blockIdx.y];
		float globalweight = d_globalweights[blockIdx.y];

		for (uint id = blockIdx.x * blockDim.x + threadIdx.x; id < elementsproj; id += gridDim.x * TpB)
		{
			uint idx = id % dimft;
			uint idy = (ndims == 3 ? id % slice : id) / dimft;
			uint idz = ndims == 3 ? id / slice : 0;

			int x = idx;
			int y = idy <= dimproj / 2 ? idy : (int)idy - (int)dimproj;
			int z = idz <= dimproj / 2 ? idz : (int)idz - (int)dimproj;
			int r2 = ndims == 3 ? z * z + y * y + x * x : y * y + x * x;
			if (r2 > rmax2)
			{
				d_proj[id] = make_hipComplex(0, 0);
				continue;
			}

			tcomplex val;
			glm::vec3 pos = glm::vec3(x, y, z);

			pos = rotation * pos;

			// Only asymmetric half is stored
			float is_neg_x = 1.0f;
			if (pos.x < -1e-5f)
			{
				// Get complex conjugated hermitian symmetry pair
				pos.x = abs(pos.x);
				pos.y = -pos.y;
				pos.z = -pos.z;
				is_neg_x = -1.0f;
			}

			// Trilinear interpolation (with physical coords)
			x0 = floor(pos.x + 1e-5f);
			x1 = x0 + 1;
			pos.x -= floor(pos.x + 1e-5f);

			y0 = floor(pos.y);
			y1 = y0 + 1;
			if (y0 < 0)
				y0 += dimvolume;
			if (y1 < 0)
				y1 += dimvolume;
			pos.y -= floor(pos.y);

			z0 = floor(pos.z);
			z1 = z0 + 1;
			if (z0 < 0)
				z0 += dimvolume;
			if (z1 < 0)
				z1 += dimvolume;
			pos.z -= floor(pos.z);

			d000 = d_volume[(z0 * dimvolume + y0) * (dimvolume / 2 + 1) + x0];
			d001 = d_volume[(z0 * dimvolume + y0) * (dimvolume / 2 + 1) + x1];
			d010 = d_volume[(z0 * dimvolume + y1) * (dimvolume / 2 + 1) + x0];
			d011 = d_volume[(z0 * dimvolume + y1) * (dimvolume / 2 + 1) + x1];
			d100 = d_volume[(z1 * dimvolume + y0) * (dimvolume / 2 + 1) + x0];
			d101 = d_volume[(z1 * dimvolume + y0) * (dimvolume / 2 + 1) + x1];
			d110 = d_volume[(z1 * dimvolume + y1) * (dimvolume / 2 + 1) + x0];
			d111 = d_volume[(z1 * dimvolume + y1) * (dimvolume / 2 + 1) + x1];

			dx00 = lerp(d000, d001, pos.x);
			dx01 = lerp(d010, d011, pos.x);
			dx10 = lerp(d100, d101, pos.x);
			dx11 = lerp(d110, d111, pos.x);

			dxy0 = lerp(dx00, dx01, pos.y);
			dxy1 = lerp(dx10, dx11, pos.y);

			val = lerp(dxy0, dxy1, pos.z);

			val.y *= is_neg_x;

			float phase = ndims == 3 ? -(x * shift.x + y * shift.y + z * shift.z) : -(x * shift.x + y * shift.y);
			val = cmul(val, make_float2(__cosf(phase), __sinf(phase)));

			val *= globalweight;

			d_proj[id] = val;
		}
	}

	template<uint ndims, uint TpB> __global__ void ProjectShifted3DtoNDKernel(cudaTex t_volumeRe, 
																				cudaTex t_volumeIm, 
																				uint dimvolume, 
																				tcomplex* d_proj, 
																				uint dimproj, 
																				size_t elementsproj, 
																				glm::mat3* d_rotations,
																				tfloat3* d_shifts,
																				float* d_globalweights,
																				uint rmax, 
																				int rmax2)
	{
		d_proj += elementsproj * blockIdx.y;

		int x0, x1, y0, y1, z0, z1;
		tcomplex d000, d010, d100, d110, d001, d011, d101, d111, dx00, dx10, dxy0, dx01, dx11, dxy1;

		uint slice = ElementsFFT1(dimproj) * dimproj;
		uint dimft = ElementsFFT1(dimproj);

		glm::mat3 rotation = d_rotations[blockIdx.y];
		tfloat3 shift = d_shifts[blockIdx.y];
		float globalweight = d_globalweights[blockIdx.y];

		for (uint id = blockIdx.x * blockDim.x + threadIdx.x; id < elementsproj; id += gridDim.x * TpB)
		{
			uint idx = id % dimft;
			uint idy = (ndims == 3 ? id % slice : id) / dimft;
			uint idz = ndims == 3 ? id / slice : 0;

			int x = idx;
			int y = idy <= dimproj / 2 ? idy : (int)idy - (int)dimproj;
			int z = idz <= dimproj / 2 ? idz : (int)idz - (int)dimproj;
			int r2 = ndims == 3 ? z * z + y * y + x * x : y * y + x * x;
			if (r2 > rmax2)
			{
				d_proj[id] = make_hipComplex(0, 0);
				continue;
			}

			tcomplex val;
			glm::vec3 pos = glm::vec3(x, y, z);

			pos = rotation * pos;

			// Only asymmetric half is stored
			float is_neg_x = 1.0f;
			if (pos.x < -1e-5f)
			{
				// Get complex conjugated hermitian symmetry pair
				pos.x = abs(pos.x);
				pos.y = -pos.y;
				pos.z = -pos.z;
				is_neg_x = -1.0f;
			}

			// Trilinear interpolation (with physical coords)
			x0 = floor(pos.x + 1e-5f);
			pos.x -= x0;
			x0 += 0.5f;
			x1 = x0 + 1.0f;

			y0 = floor(pos.y);
			pos.y -= y0;
			y1 = y0 + 1;
			if (y0 < 0)
				y0 += dimvolume;
			y0 += 0.5f;
			if (y1 < 0)
				y1 += dimvolume;
			y1 += 0.5f;

			z0 = floor(pos.z);
			pos.z -= z0;
			z1 = z0 + 1;
			if (z0 < 0)
				z0 += dimvolume;
			z0 += 0.5f;
			if (z1 < 0)
				z1 += dimvolume;
			z1 += 0.5f;

			d000 = make_hipComplex(tex3D<tfloat>(t_volumeRe, x0, y0, z0), tex3D<tfloat>(t_volumeIm, x0, y0, z0));
			d001 = make_hipComplex(tex3D<tfloat>(t_volumeRe, x1, y0, z0), tex3D<tfloat>(t_volumeIm, x1, y0, z0));
			d010 = make_hipComplex(tex3D<tfloat>(t_volumeRe, x0, y1, z0), tex3D<tfloat>(t_volumeIm, x0, y1, z0));
			d011 = make_hipComplex(tex3D<tfloat>(t_volumeRe, x1, y1, z0), tex3D<tfloat>(t_volumeIm, x1, y1, z0));
			d100 = make_hipComplex(tex3D<tfloat>(t_volumeRe, x0, y0, z1), tex3D<tfloat>(t_volumeIm, x0, y0, z1));
			d101 = make_hipComplex(tex3D<tfloat>(t_volumeRe, x1, y0, z1), tex3D<tfloat>(t_volumeIm, x1, y0, z1));
			d110 = make_hipComplex(tex3D<tfloat>(t_volumeRe, x0, y1, z1), tex3D<tfloat>(t_volumeIm, x0, y1, z1));
			d111 = make_hipComplex(tex3D<tfloat>(t_volumeRe, x1, y1, z1), tex3D<tfloat>(t_volumeIm, x1, y1, z1));

			dx00 = lerp(d000, d001, pos.x);
			dx01 = lerp(d010, d011, pos.x);
			dx10 = lerp(d100, d101, pos.x);
			dx11 = lerp(d110, d111, pos.x);

			dxy0 = lerp(dx00, dx01, pos.y);
			dxy1 = lerp(dx10, dx11, pos.y);

			val = lerp(dxy0, dxy1, pos.z);

			val.y *= is_neg_x;

			float phase = ndims == 3 ? -(x * shift.x + y * shift.y + z * shift.z) : -(x * shift.x + y * shift.y);
			val = cmul(val, make_float2(__cosf(phase), __sinf(phase)));

			val *= globalweight;

			d_proj[id] = val;
		}
	}
}
