#include "hip/hip_runtime.h"
#include "gtom/include/Prerequisites.cuh"

namespace gtom
{
	template<bool docc> __global__ void DiffRemapDenseKernel(tfloat* d_input, tfloat* d_output, uint3* d_orientationindices, uint elements, uint iclass, uint nparticles, uint nclasses, uint nrot, uint ntrans, uint ntranspadded, const tfloat* __restrict__ d_xi2imgs, const tfloat* __restrict__ d_sqrtxi2);
	template<bool docc> __global__ void DiffRemapSparseKernel(tfloat* d_input, tfloat* d_output, tfloat* d_mindiff2, const uint3* __restrict__ d_combinations, uint* d_hiddenover, uint elements, uint tileelements, uint weightsperpart, uint nparticles, const tfloat* __restrict__ d_xi2imgs, const tfloat* __restrict__ d_sqrtxi2);

	void d_rlnDiffRemapDense(tfloat* d_input, tfloat* d_output, uint3* d_orientationindices, uint norientations, uint iclass, uint nparticles, uint nclasses, uint nrot, uint ntrans, uint ntranspadded, tfloat* d_xi2imgs, tfloat* d_sqrtxi2, bool docc)
	{
		uint elements = norientations * nparticles * ntrans;
		uint TpB = 128;
		dim3 grid = dim3((elements + TpB - 1) / TpB, 1, 1);
		if (docc)
			DiffRemapDenseKernel<true> <<<grid, TpB>>> (d_input, d_output, d_orientationindices, elements, iclass, nparticles, nclasses, nrot, ntrans, ntranspadded, d_xi2imgs, d_sqrtxi2);
		else
			DiffRemapDenseKernel<false> <<<grid, TpB>>> (d_input, d_output, d_orientationindices, elements, iclass, nparticles, nclasses, nrot, ntrans, ntranspadded, d_xi2imgs, d_sqrtxi2);
	}

	void d_rlnDiffRemapSparse(tfloat* d_input, tfloat* d_output, tfloat* d_mindiff2, uint3* d_combinations, uint* d_hiddenover, uint elements, uint tileelements, uint weightsperpart, uint nparticles, tfloat* d_xi2imgs, tfloat* d_sqrtxi2, bool docc)
	{
		uint TpB = 128;
		dim3 grid = dim3((elements + TpB - 1) / TpB, 1, 1);
		if (docc)
			DiffRemapSparseKernel<true> << <grid, TpB >> > (d_input, d_output, d_mindiff2, d_combinations, d_hiddenover, elements, tileelements, weightsperpart, nparticles, d_xi2imgs, d_sqrtxi2);
		else
			DiffRemapSparseKernel<false> << <grid, TpB >> > (d_input, d_output, d_mindiff2, d_combinations, d_hiddenover, elements, tileelements, weightsperpart, nparticles, d_xi2imgs, d_sqrtxi2);
	}

	template<bool docc> __global__ void DiffRemapDenseKernel(tfloat* d_input, tfloat* d_output, uint3* d_orientationindices, uint elements, uint iclass, uint nparticles, uint nclasses, uint nrot, uint ntrans, uint ntranspadded, const tfloat* __restrict__ d_xi2imgs, const tfloat* __restrict__ d_sqrtxi2)
	{
		for (uint id = blockIdx.x * blockDim.x + threadIdx.x; id < elements; id += gridDim.x * blockDim.x)
		{
			uint irot = id / (nparticles * ntrans);
			uint ipart = (id - irot * nparticles * ntrans) / ntrans;
			uint itrans = id % ntrans;
			uint iorient = d_orientationindices[irot].y;

			size_t idinput = (irot * nparticles + ipart) * ntranspadded + itrans;
			size_t idoutput = ((ipart * nclasses + iclass) * nrot + iorient) * ntrans + itrans;

			tfloat val = d_input[idinput];
			if (docc)
				val *= d_sqrtxi2[ipart];	// Already 1/x
			else
				val += d_xi2imgs[ipart];	// Already x/2

			d_output[idoutput] = val;
		}
	}

	__device__ inline void fatomicMin(tfloat* addr, tfloat val)
	{
		int* address_as_i = (int*)addr;
		int old = *address_as_i, assumed;
		do 
		{
			assumed = old;
			old = atomicCAS(address_as_i, assumed, __float_as_int(fminf(val, __int_as_float(assumed))));
		} 
		while (assumed != old);
	}

	template<bool docc> __global__ void DiffRemapSparseKernel(tfloat* d_input, tfloat* d_output, tfloat* d_mindiff2, const uint3* __restrict__ d_combinations, uint* d_hiddenover, uint elements, uint tileelements, uint weightsperpart, uint nparticles, const tfloat* __restrict__ d_xi2imgs, const tfloat* __restrict__ d_sqrtxi2)
	{
		__shared__ tfloat s_mindiff2[1024];
		for (uint i = threadIdx.x; i < nparticles; i += blockDim.x)
			s_mindiff2[i] = 1e30;
		__syncthreads();

		for (uint id = blockIdx.x * blockDim.x + threadIdx.x; id < elements; id += gridDim.x * blockDim.x)
		{
			uint ipart = d_combinations[id / tileelements].y;

			tfloat val = d_input[id];
			if (docc)
				val *= d_sqrtxi2[ipart];	// Already 1/x
			else
				val += d_xi2imgs[ipart];	// Already x/2

			d_output[weightsperpart * ipart + d_hiddenover[id]] = val;
			
			fatomicMin(&s_mindiff2[ipart], val);
		}
		__syncthreads();

		for (uint i = threadIdx.x; i < nparticles; i += blockDim.x)
			fatomicMin(d_mindiff2 + i, s_mindiff2[i]);
	}
}
