#include "hip/hip_runtime.h"
#include "gtom/include/Prerequisites.cuh"


namespace gtom
{
	////////////////////////////
	//CUDA kernel declarations//
	////////////////////////////

	template<int mode, int ndims> __global__ void WindowMaskKernel(tfloat* d_input, tfloat* d_output, int3 dims, tfloat radius, tfloat3 center, int batch);
	template<int mode, int ndims> __global__ void WindowMaskBorderDistanceKernel(tfloat* d_input, tfloat* d_output, int3 dims, int falloff, int batch);


	////////////////
	//Host methods//
	////////////////

	void d_HannMask(tfloat* d_input, tfloat* d_output, int3 dims, tfloat* radius, tfloat3* center, int batch)
	{
		tfloat _radius = radius != NULL ? *radius : tmin(dims.z > 1 ? tmin(dims.x, dims.z) : dims.x, dims.y) / 2 - 1;
		tfloat3 _center = center != NULL ? *center : tfloat3(dims.x / 2, dims.y / 2, dims.z / 2);

		int TpB = tmin(NextMultipleOf(dims.x, 32), 256);
		dim3 grid = dim3(dims.y, dims.z, 1);
		if (DimensionCount(dims) == 1)
			WindowMaskKernel<0, 1> << <grid, TpB >> > (d_input, d_output, dims, _radius, _center, batch);
		if (DimensionCount(dims) == 2)
			WindowMaskKernel<0, 2> << <grid, TpB >> > (d_input, d_output, dims, _radius, _center, batch);
		if (DimensionCount(dims) == 3)
			WindowMaskKernel<0, 3> << <grid, TpB >> > (d_input, d_output, dims, _radius, _center, batch);
	}

	void d_HammingMask(tfloat* d_input, tfloat* d_output, int3 dims, tfloat* radius, tfloat3* center, int batch)
	{
		tfloat _radius = radius != NULL ? *radius : tmin(dims.z > 1 ? tmin(dims.x, dims.z) : dims.x, dims.y) / 2 - 1;
		tfloat3 _center = center != NULL ? *center : tfloat3(dims.x / 2, dims.y / 2, dims.z / 2);

		int TpB = tmin(NextMultipleOf(dims.x, 32), 256);
		dim3 grid = dim3(dims.y, dims.z, 1);
		if (DimensionCount(dims) == 1)
			WindowMaskKernel<1, 1> << <grid, TpB >> > (d_input, d_output, dims, _radius, _center, batch);
		if (DimensionCount(dims) == 2)
			WindowMaskKernel<1, 2> << <grid, TpB >> > (d_input, d_output, dims, _radius, _center, batch);
		if (DimensionCount(dims) == 3)
			WindowMaskKernel<1, 3> << <grid, TpB >> > (d_input, d_output, dims, _radius, _center, batch);
	}

	void d_GaussianMask(tfloat* d_input, tfloat* d_output, int3 dims, tfloat* sigma, tfloat3* center, int batch)
	{
		tfloat _sigma = sigma != NULL ? *sigma : (tfloat)1;
		tfloat3 _center = center != NULL ? *center : tfloat3(dims.x / 2, dims.y / 2, dims.z / 2);

		int TpB = tmin(NextMultipleOf(dims.x, 32), 256);
		dim3 grid = dim3(dims.y, dims.z, 1);
		if (DimensionCount(dims) == 1)
			WindowMaskKernel<2, 1> << <grid, TpB >> > (d_input, d_output, dims, (tfloat)2 * _sigma * _sigma, _center, batch);
		if (DimensionCount(dims) == 2)
			WindowMaskKernel<2, 2> << <grid, TpB >> > (d_input, d_output, dims, (tfloat)2 * _sigma * _sigma, _center, batch);
		if (DimensionCount(dims) == 3)
			WindowMaskKernel<2, 3> << <grid, TpB >> > (d_input, d_output, dims, (tfloat)2 * _sigma * _sigma, _center, batch);
	}

	void d_HannMaskBorderDistance(tfloat* d_input, tfloat* d_output, int3 dims, int falloff, int batch)
	{
		int TpB = tmin(NextMultipleOf(dims.x, 32), 256);
		dim3 grid = dim3(dims.y, dims.z, 1);
		if (DimensionCount(dims) == 1)
			WindowMaskBorderDistanceKernel<0, 1> << <grid, TpB >> > (d_input, d_output, dims, falloff, batch);
		if (DimensionCount(dims) == 2)
			WindowMaskBorderDistanceKernel<0, 2> << <grid, TpB >> > (d_input, d_output, dims, falloff, batch);
		if (DimensionCount(dims) == 3)
			WindowMaskBorderDistanceKernel<0, 3> << <grid, TpB >> > (d_input, d_output, dims, falloff, batch);
	}

	void d_HammingMaskBorderDistance(tfloat* d_input, tfloat* d_output, int3 dims, int falloff, int batch)
	{
		int TpB = tmin(NextMultipleOf(dims.x, 32), 256);
		dim3 grid = dim3(dims.y, dims.z, 1);
		if (DimensionCount(dims) == 1)
			WindowMaskBorderDistanceKernel<1, 1> << <grid, TpB >> > (d_input, d_output, dims, falloff, batch);
		if (DimensionCount(dims) == 2)
			WindowMaskBorderDistanceKernel<1, 2> << <grid, TpB >> > (d_input, d_output, dims, falloff, batch);
		if (DimensionCount(dims) == 3)
			WindowMaskBorderDistanceKernel<1, 3> << <grid, TpB >> > (d_input, d_output, dims, falloff, batch);
	}

	////////////////
	//CUDA kernels//
	////////////////

	template<int mode, int ndims> __global__ void WindowMaskKernel(tfloat* d_input, tfloat* d_output, int3 dims, tfloat radius, tfloat3 center, int batch)
	{
		tfloat xsq, ysq, zsq, length;

		if (ndims > 1)
		{
			ysq = (tfloat)blockIdx.x - center.y;
			ysq *= ysq;
		}
		else
			ysq = 0;

		if (ndims > 2)
		{
			zsq = (tfloat)blockIdx.y - center.z;
			zsq *= zsq;
		}
		else
			zsq = 0;

		for (int x = threadIdx.x; x < dims.x; x += blockDim.x)
		{
			xsq = (tfloat)x - center.x;
			xsq *= xsq;

			length = sqrt(xsq + ysq + zsq);

			tfloat val = 0;
			//Hann
			if (mode == 0)
				val = (tfloat)0.5 * ((tfloat)1 + cos(min(length / radius, (tfloat)1) * PI));
			//Hamming
			else if (mode == 1)
				val = (tfloat)0.54 - (tfloat)0.46 * cos(((tfloat)1 - min(length / radius, (tfloat)1)) * PI);
			//Gaussian
			else if (mode == 2)
				val = exp(-(pow(length, (tfloat)2) / radius));

			for (int b = 0; b < batch; b++)
			{
				if (ndims > 2)
					d_output[Elements(dims) * b + (blockIdx.y * dims.y + blockIdx.x) * dims.x + x] = val * d_input[Elements(dims) * b + (blockIdx.y * dims.y + blockIdx.x) * dims.x + x];
				else
					d_output[Elements(dims) * b + blockIdx.x * dims.x + x] = val * d_input[Elements(dims) * b + blockIdx.x * dims.x + x];
			}
		}
	}

	template<int mode, int ndims> __global__ void WindowMaskBorderDistanceKernel(tfloat* d_input, tfloat* d_output, int3 dims, int falloff, int batch)
	{
		int distx = 0, disty = 0, distz = 0;

		if (ndims > 1)
		{
			int y = blockIdx.x;
			int fromtop = max(0, falloff - y);
			int frombottom = max(0, falloff - (dims.y - 1 - y));
			disty = max(fromtop, frombottom);
		}

		if (ndims > 2)
		{
			int z = blockIdx.y;
			int fromback = max(0, falloff - z);
			int fromfront = max(0, falloff - (dims.z - 1 - z));
			distz = max(fromback, fromfront);
		}

		for (int idx = threadIdx.x; idx < dims.x; idx += blockDim.x)
		{
			int fromleft = max(0, falloff - idx);
			int fromright = max(0, falloff - (dims.x - 1 - idx));
			distx = max(fromleft, fromright);

			float dist;
			if (ndims == 3)
				dist = sqrt(float(distx * distx + disty * disty + distz * distz));
			else if (ndims == 2)
				dist = sqrt(float(distx * distx + disty * disty));
			else
				dist = (float)distx;


			tfloat val = 0;
			//Hann
			if (mode == 0)
			{
				val = 0.5f * (1.0f + cos(min(dist / (float)falloff, 1.0f) * PI));
			}
			//Hamming
			else if (mode == 1)
			{
				val = 0.54f - 0.46f * cos((1.0f - min(dist / (float)falloff, 1.0f)) * PI);
			}

			for (int b = 0; b < batch; b++)
			{
				if (ndims == 3)
					d_output[Elements(dims) * b + (blockIdx.y * dims.y + blockIdx.x) * dims.x + idx] = val * d_input[Elements(dims) * b + (blockIdx.y * dims.y + blockIdx.x) * dims.x + idx];
				else
					d_output[Elements(dims) * b + blockIdx.x * dims.x + idx] = val * d_input[Elements(dims) * b + blockIdx.x * dims.x + idx];
			}
		}
	}
}