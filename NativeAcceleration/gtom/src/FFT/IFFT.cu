#include "hip/hip_runtime.h"
#include "gtom/include/Prerequisites.cuh"
#include "gtom/include/FFT.cuh"
#include "gtom/include/Generics.cuh"
#include "gtom/include/Helper.cuh"

namespace gtom
{
	void d_IFFTC2R(tcomplex* const d_input, tfloat* const d_output, int const ndimensions, int3 const dimensions, int batch, bool renormalize)
	{
		hipfftHandle plan = d_IFFTC2RGetPlan(ndimensions, dimensions, batch);
		if (renormalize)
			d_IFFTC2R(d_input, d_output, &plan, dimensions, batch);
		else
			d_IFFTC2R(d_input, d_output, &plan);
		hipfftDestroy(plan);
	}

	hipfftHandle d_IFFTC2RGetPlan(int const ndimensions, int3 const dimensions, int batch)
	{
		hipfftHandle plan;
		hipfftType direction = IS_TFLOAT_DOUBLE ? HIPFFT_Z2D : HIPFFT_C2R;
		int n[3] = { dimensions.z, dimensions.y, dimensions.x };

		CHECK_CUFFT_ERRORS(hipfftPlanMany(&plan, ndimensions, n + (3 - ndimensions),
										 NULL, 1, 0,
										 NULL, 1, 0,
										 direction, batch));

		//cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE);

		hipfftSetStream(plan, hipStreamDefault);

		return plan;
	}

	void d_IFFTC2R(tcomplex* const d_input, tfloat* const d_output, hipfftHandle* plan, int3 dimensions, int batch)
	{
#ifdef GTOM_DOUBLE
		hipfftExecZ2D(*plan, d_input, d_output);
#else
		CHECK_CUFFT_ERRORS(hipfftExecC2R(*plan, d_input, d_output));
#endif
		hipStreamSynchronize(hipStreamDefault);

		d_MultiplyByScalar(d_output, d_output, Elements(dimensions) * batch, 1.0f / (float)Elements(dimensions));
	}

	void d_IFFTC2R(tcomplex* const d_input, tfloat* const d_output, hipfftHandle* plan)
	{
#ifdef GTOM_DOUBLE
		hipfftExecZ2D(*plan, d_input, d_output);
#else
		hipfftExecC2R(*plan, d_input, d_output);
#endif
		hipStreamSynchronize(hipStreamDefault);
	}

	void d_IFFTZ2D(hipfftDoubleComplex* const d_input, double* const d_output, int const ndimensions, int3 const dimensions, int batch)
	{
		hipfftHandle plan;
		hipfftType direction = HIPFFT_Z2D;
		int n[3] = { dimensions.z, dimensions.y, dimensions.x };

		hipfftPlanMany(&plan, ndimensions, n + (3 - ndimensions),
			NULL, 1, 0,
			NULL, 1, 0,
			direction, batch);

		//cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE);
#ifdef GTOM_DOUBLE
		hipfftExecZ2D(plan, d_input, d_output);
#else
		hipfftExecZ2D(plan, d_input, d_output);
#endif

		hipfftDestroy(plan);

		size_t elements = dimensions.x * dimensions.y * dimensions.z;
		d_MultiplyByScalar(d_output, d_output, elements, 1.0 / (double)elements);
	}

	void d_IFFTC2RFull(tcomplex* const d_input, tfloat* const d_output, int const ndimensions, int3 const dimensions, int batch)
	{
		tcomplex* d_complexoutput;
		hipMalloc((void**)&d_complexoutput, Elements(dimensions) * sizeof(tcomplex));

		d_IFFTC2C(d_input, d_complexoutput, ndimensions, dimensions, batch);
		d_Re(d_complexoutput, d_output, Elements(dimensions));

		hipFree(d_complexoutput);
	}

	void d_IFFTC2C(tcomplex* const d_input, tcomplex* const d_output, int const ndimensions, int3 const dimensions, int batch)
	{
		hipfftHandle plan = d_IFFTC2CGetPlan(ndimensions, dimensions, batch);
		d_IFFTC2C(d_input, d_output, &plan, dimensions);
		hipfftDestroy(plan);
	}

	hipfftHandle d_IFFTC2CGetPlan(int const ndimensions, int3 const dimensions, int batch)
	{
		hipfftHandle plan;
		hipfftType direction = IS_TFLOAT_DOUBLE ? HIPFFT_Z2Z : HIPFFT_C2C;
		int n[3] = { dimensions.z, dimensions.y, dimensions.x };

		hipfftPlanMany(&plan, ndimensions, n + (3 - ndimensions),
			NULL, 1, 0,
			NULL, 1, 0,
			direction, batch);

		//cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE);

		return plan;
	}

	void d_IFFTC2C(tcomplex* const d_input, tcomplex* const d_output, hipfftHandle* plan, int3 const dimensions)
	{
#ifdef GTOM_DOUBLE
		hipfftExecZ2Z(*plan, d_input, d_output);
#else
		hipfftExecC2C(*plan, d_input, d_output, HIPFFT_BACKWARD);
#endif
		hipStreamQuery(0);

		size_t elements = dimensions.x * dimensions.y * dimensions.z;
		d_MultiplyByScalar((tfloat*)d_output, (tfloat*)d_output, elements * 2, 1.0f / (float)elements);
	}
}